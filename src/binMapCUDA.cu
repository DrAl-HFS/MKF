#include "hip/hip_runtime.h"
// binMapCUDA.h - packed binary map generation from scalar fields
// https://github.com/DrAl-HFS/MKF.git
// (c) Project Contributors Jan-Sept 2019

#include "binMapCUDA.h"
#include "utilCUDA.hpp"


#define VT_WRDS 5
#define VT_WRDN (1<<VT_WRDS)
#define VT_WRDM (VT_WRDN-1)

#define VT_BLKS 7             // 7
#define VT_BLKN (1<<VT_BLKS)
#define VT_BLKM (VT_BLKN-1)

#define VT_BWS (VT_BLKS - VT_WRDS)


/***/

struct CUDAFieldDesc
{  // Expect multiple fields, common def & stride
   int            nF;
   FieldDef       def[3];
   FieldStride    stride[3];
   ConstFieldPtr  field[BMFI_FIELD_MAX];
};

// Refactor to CTOR + check ? ...
static int checkFD (CUDAFieldDesc *pD, const BMFieldInfo *pI)
{
   if (pD && pI && pI->pD)
   {
      int pad= (pI->pD[0] & VT_WRDM);
      int n=0;
      if (0 != pI->fieldMask)
      {
         for (int i=0; i<BMFI_FIELD_MAX; i++)
         {
            pD->field[n]= pI->field[i];
            n+= (NULL != pI->field[i].p);
         }
         //if (n != m) WARN();
         pD->nF= n;
         if (pI->pS)
         {  // Validate stride ?
            for (int i=0; i<3; i++) { pD->def[i]= pI->pD[i]; pD->stride[i]= pI->pS[i]; }
         }
         else
         {  // Generate stride
            FieldStride k= 1;
            for (int i=0; i<3; i++)
            {
               pD->def[i]= pI->pD[i];
               pD->stride[i]= k;
               k*= pD->def[i];
            }
         }
         if (n > 1) { return(4); } // if (pad) WARN();
         else { return( 1 + (0 != pad) ); }
      }
   }
   return(0);
} // checkFD

/***/

template <typename T_Elem>
class CUDAMap
{
protected:
   T_Elem t[BM_NUMT];
   uint     m;

public:
   CUDAMap (const BinMapF32 *pM) { m= pM->m; for (int i=0; i<BM_NUMT; i++) { t[i]= pM->t[i]; } }
   CUDAMap (const BinMapF64 *pM) { m= pM->m; for (int i=0; i<BM_NUMT; i++) { t[i]= pM->t[i]; } }

   __device__ uint eval (const T_Elem f) const
   {
      const uint d= (1 + (f > t[0]) - (f < t[0]));
      return( (m >> d) & 0x1 );
   } // eval
   __device__ uint operator () (const T_Elem f) const { eval(f); }
}; // template class CUDAMap

template <typename T_Elem>
class CUDAFieldMap : protected CUDAMap<T_Elem>
{
protected:
   const T_Elem *pF;

public:
   CUDAFieldMap (const T_Elem * p, const BinMapF32 *pM) : CUDAMap<T_Elem>(pM) { pF= p; }
   CUDAFieldMap (const T_Elem * p, const BinMapF64 *pM) : CUDAMap<T_Elem>(pM) { pF= p; }

   //__device__ uint operator () (const size_t i) const { return CUDAMap<T_Elem>::operator () (pF[i]); }
   __device__ uint operator () (const size_t i) const { return CUDAMap<T_Elem>::eval(pF[i]); }
}; // template class CUDAFieldMap

template <typename T_Elem>
class CUDAMultiField
{  // Multiple fields with common stride
protected:
   const T_Elem   * fPtr[BMFI_FIELD_MAX];
   FieldStride stride[3];
   FieldDef    def[3];
   uint        nF;

   void setDS (const FieldDef d[3], const FieldStride *pS)
   {
      if (pS)
      {  // copy
         for (int i=0; i<3; i++) { def[i]= d[i]; stride[i]= pS[i]; }
      }
      else
      {  // Generate stride
         FieldStride k= 1;
         for (int i=0; i<3; i++)
         {
            def[i]= d[i];
            stride[i]= k;
            k*= def[i];
         }
      }
   } // setDS

   uint setF (const ConstFieldPtr a[], const uint m)
   {
      uint n= 0;
      for (int i=0; i<BMFI_FIELD_MAX; i++)
      {
         if ((NULL != a[i].p) && (m & (0x1 << i)))
         {
            fPtr[n++]= (const T_Elem*) a[i].p;
         }
      }
      return(n);
   } // setF

public:
   CUDAMultiField (const BMFieldInfo *pI)
   {
      if (pI->pD)
      {
         setDS(pI->pD, pI->pS);
         nF= setF(pI->field, pI->fieldMask);
      }
   } // CTOR

   __device__ size_t index (uint x, uint y, uint z) const { return(x * stride[0] + y * stride[1] + z * stride[2]); }

   __device__ T_Elem sum (const size_t i) const
   {
      T_Elem s= (fPtr[0])[i];
      for (int iF=1; iF < nF; iF++) { s+= (fPtr[iF])[i]; }
      return(s);
   } // sum
   __device__ T_Elem operator () (const size_t i) const { return sum(i); }
}; // template class CUDAMultiField

template <typename T_Elem>
class CUDAMultiFieldMap : protected CUDAMultiField<T_Elem>, CUDAMap<T_Elem>
{
public:
   CUDAMultiFieldMap (const BMFieldInfo *pI, const BinMapF32 *pM) : CUDAMultiField<T_Elem>(pI), CUDAMap<T_Elem>(pM) {;}
   CUDAMultiFieldMap (const BMFieldInfo *pI, const BinMapF64 *pM) : CUDAMultiField<T_Elem>(pI), CUDAMap<T_Elem>(pM) {;}

   __device__ size_t index (uint x, uint y, uint z) const { return CUDAMultiField<T_Elem>::index(x,y,z); }
   __device__ uint operator () (const size_t i) const { return CUDAMap<T_Elem>::eval( CUDAMultiField<T_Elem>::sum(i) ); }
}; // template class CUDAMultiFieldMap


/* DEPRECATION ZONE */

__device__ int bm1f32 (const float f, const BinMapF32& bm)
{
   const int d= (1 + (f > bm.t[0]) - (f < bm.t[0]));
   return( (bm.m >> d) & 0x1 );
} // bm1f32

/*

__global__ void vThreshV32
(
   BMPackWord rBM[],
   const CUDAFieldDesc fd,
   const BMOrg bmo,
   const BinMapF32 bm
)
{
   const uint x= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ BMPackWord w[VT_BLKN];
   if (x < fd.def[0])
   {
      size_t i= x * fd.stride[0] + blockIdx.y * fd.stride[1] + blockIdx.z * fd.stride[2];
      const uint j= threadIdx.x & VT_WRDM;
      const uint k= threadIdx.x & ~VT_WRDM;

      w[threadIdx.x]= bm1f32( fd.field[0].pF32[i], bm ) << j;

      if (0 == merge32(w+k, j)) // & VT_WRDM) if BLKS > WRDS !
      {  // (x >> VT_WRDS)
         i= blockIdx.x + (k >> VT_WRDS) + blockIdx.y * bmo.rowWS + blockIdx.z * bmo.planeWS;
         rBM[i]= w[threadIdx.x];
      }
   }
} // vThreshV32

__global__ void vThresh8 (uint r[], const float f[], const size_t n, const BinMapF32 mc)
{
   const size_t i= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ uint z[VT_BLKN];
   if (i < n)
   {
      const int j= i & VT_BLKM;
      const int k= i & 0x7; // j & 0x7
      const int d= 1 + (f[i] > mc.t[0]) - (f[i] < mc.t[0]);
      z[j]= ((mc.m >> d) & 0x1) << k; // smaller shift faster ?

      __syncthreads();

      if (0 == k)
      {  // j : { 0, 8, 16, 24 } 4P, 7I
         for (int l=1; l<8; l++) { z[j]|= z[j+l]; }

         __syncthreads();

         if (0 == j)
         {
            r[i>>VT_BLKS]= ( z[0] << 0 ) | ( z[8] << 8 ) | ( z[16] << 16 ) | ( z[24] << 24 );
         }
      }
   }
} // vThresh8

__global__ void vThreshL32 (BMPackWord r[], const float f[], const size_t n, const BinMapF32 bm)
{
   const size_t i= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ uint z[VT_BLKN];
   if (i < n)
   {
      const int j= i & VT_BLKM;

      z[j]= bm1f32(f[i],bm) << j;

      merge32(z, j);
      if (0 == j) { r[i>>VT_WRDS]= z[0] | z[16]; }
/ *
      __syncthreads();

      if (0 == (j & 0x3))
      {  // j : { 0, 4, 8, 12, 16, 10, 24, 28 } 8P 3I
         for (int l=1; l<4; l++) { z[j]|= z[j+l]; }

         __syncthreads();

         //if (0 == j) { r[i>>BLKS]= z[0] | z[4] | z[8] | z[12] | z[16] | z[20] | z[24] | z[28]; }
         if (0 == (j & 0xF))
         {  // j : { 0, 16 } 2P 3I
            for (int l=4; l<16; l+=4) { z[j]|= z[j+l]; }

            __syncthreads();

            if (0 == j) { r[i>>VT_BLKS]= z[0] | z[16]; }
         }
      }
* /
   }
} // vThreshL32

*/

/***/

// local mem bit merge util
__device__ int merge32 (BMPackWord w[32], const int j)
{
/* TODO: consider using CUDA9 warp level primitives...
#define FULL_MASK 0xffffffff
for (int offset = 16; offset > 0; offset /= 2)
    val += __shfl_down_sync(FULL_MASK, val, offset);
*/
   //__syncthreads(); // Unnecessary - no divergence at this point

   if (0 == (j & 0x3))
   {  // j : { 0, 4, 8, 12, 16, 10, 24, 28 } 8P 3I
      for (int l=1; l<4; l++) { w[j]|= w[j+l]; }

      __syncthreads(); // Required for (unexplained) divergence

      if (0 == (j & 0xF))
      {  // j : { 0, 16 } 2P 3I
         for (int l=4; l<16; l+=4) { w[j]|= w[j+l]; }

         __syncthreads(); //  Optional ?
         if (0 == j) { w[0]|= w[16]; }
      }
   }
   return(j);
} // merge32

/***/

template <typename T_Elem>
__global__ void mapFieldL32 (BMPackWord rBM[], const CUDAFieldMap<T_Elem> f, const size_t n)
{
   const size_t i= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ BMPackWord w[VT_BLKN];
   if (i < n)
   {
      const uint j= threadIdx.x & VT_WRDM;   // lane index (bit number)
      const uint k= threadIdx.x & ~VT_WRDM;  // warp index (word number)

      w[threadIdx.x]= f(i) << j;

      if (0 == merge32(w+k, j)) { rBM[i>>VT_WRDS]= w[threadIdx.x]; }
   }
} // mapFieldL32

template <typename T_Elem>
__global__ void mapFieldV32 (BMPackWord rBM[], const BMOrg bmo, const CUDAMultiFieldMap<T_Elem> f) // const size_t n)
{
   const uint x= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ BMPackWord w[VT_BLKN];
   if (x < bmo.rowElem)
   {
      size_t i= f.index(x, blockIdx.y, blockIdx.z);
      const uint j= threadIdx.x & VT_WRDM;
      const uint k= threadIdx.x & ~VT_WRDM;

      w[threadIdx.x]= f(i) << j;

      if (0 == merge32(w+k, j))
      {  // One thread per word
         i= ((blockIdx.x << VT_BWS) + (k >> VT_WRDS)) +
               blockIdx.y * bmo.rowWS + blockIdx.z * bmo.planeWS;
         rBM[i]= w[threadIdx.x];
      }
   }
} // mapFieldV32

__global__ void vThreshVSum32
(
   BMPackWord rBM[],
   const CUDAFieldDesc fd,
   const BMOrg bmo,
   const BinMapF32 bm
)
{
   const uint x= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ BMPackWord w[VT_BLKN];
   if (x < bmo.rowElem)// && (y < defY))
   {
      size_t i= x * fd.stride[0] + blockIdx.y * fd.stride[1] + blockIdx.z * fd.stride[2];
      const uint j= threadIdx.x & VT_WRDM;
      const uint k= threadIdx.x & ~VT_WRDM;

      float s= (fd.field[0].pF32)[i];
      for (int f=1; f < fd.nF; f++) { s+= fd.field[f].pF32[i]; }

      w[threadIdx.x]= bm1f32(s,bm) << j;

      if (0 == merge32(w, j))
      {
         i= ((blockIdx.x << VT_BWS) + (k >> VT_WRDS)) +
               blockIdx.y * bmo.rowWS + blockIdx.z * bmo.planeWS;
         rBM[i]= w[threadIdx.x];
      }
   }
} // vThreshVSum32


/* INTERFACE */

extern "C"
BMOrg *binMapCUDA
(
   BMPackWord  * pW,
   BMOrg       * pO,
   const BMFieldInfo * pF,
   const BinMapF32   * pMC
)
{
   CUDAFieldDesc fd;
   const int id= checkFD(&fd, pF);
   if (id > 0)
   {
      CTimerCUDA t;
      const char * pID= NULL;
      const int   nBlkRow= (fd.def[0] + VT_BLKM) / VT_BLKN;

      if ( setBMO(pO, fd.def, pF->profID) )
      {
         const dim3 grd(nBlkRow, fd.def[1], fd.def[2]);
         const dim3 blk(VT_BLKN,1,1);
         switch (pF->profID & 0x30)
         {
            case 0x10 :
            if (0 == (fd.def[0] & VT_BLKM)) // 1D collapsable
            {  const size_t nF= prodNI(fd.def,3);
               mapFieldL32<<<nF/VT_BLKN,VT_BLKN>>>(pW, CUDAFieldMap<float>(fd.field[0].pF32, pMC), nF);
               pID= "mapFieldL32()"; // "vThreshL32()";
               break;
            }
            //else...
            case 0x00 : // Horribly inefficient iteration - only method presently working for !=*32 row length
            {  const int nRows= prodNI(fd.def+1,2);
               for (int i=0; i<nRows; i++)
               {
                  mapFieldL32<<<nBlkRow,VT_BLKN>>>(pW + i * pO->rowWS, CUDAFieldMap<float>(fd.field[0].pF32 + i * fd.stride[1], pMC), fd.def[0]);
               }
               pID= "nRows*mapFieldL32()";
            }  break;

            case 0x20 :
               mapFieldV32<<<grd,blk>>>(pW, *pO, CUDAMultiFieldMap<float>(pF, pMC));
               pID= "mapFieldV32()";
               break;

            case 0x30 :
               vThreshVSum32<<<grd,blk>>>(pW, fd, *pO, *pMC);
               pID= "vThreshVSum32()";
               break;
         }
         LOG("binMapCUDA() - %s<<<%u>>>() - dt= %Gms\n", pID, blk.x, t.elapsedms());
         if (0 == ctuErr(NULL, pID)) { return(pO); }
      }
   }
   return(NULL);
} // binMapCUDA
