#include "hip/hip_runtime.h"
// binMapCUDA.h - packed binary map generation from scalar fields
// https://github.com/DrAl-HFS/MKF.git
// (c) Project Contributors Jan-Sept 2019

#include "binMapCUDA.h"
#include "utilCUDA.hpp"


#define VT_WRDS 5
#define VT_WRDN (1<<VT_WRDS)
#define VT_WRDM (VT_WRDN-1)

#define VT_BLKS 5
#define VT_BLKN (1<<VT_BLKS)
#define VT_BLKM (VT_BLKN-1)

/***/

struct CUDAFieldDesc
{  // Expect multiple fields, common def & stride
   int            nF;
   FieldDef       def[3];
   FieldStride    stride[3];
   ConstFieldPtr  field[BMFI_FIELD_MAX];
};

// Refactor to CTOR + check ? ...
static int checkFD (CUDAFieldDesc *pD, const BMFieldInfo *pI)
{
   if (pD && pI && pI->pD)
   {
      int pad= (pI->pD[0] & VT_WRDM);
      int n=0, m= MIN(BMFI_FIELD_MAX, pI->nField);
      if (m > 0)
      {
         for (int i=0; i<m; i++)
         {
            pD->field[n]= pI->field[i];
            n+= (NULL != pI->field[i].p);
         }
         //if (n != m) WARN();
         pD->nF= n;
         if (pI->pS)
         {  // Validate stride ?
            for (int i=0; i<3; i++) { pD->def[i]= pI->pD[i]; pD->stride[i]= pI->pS[i]; }
         }
         else
         {  // Generate stride
            FieldStride k= 1;
            for (int i=0; i<3; i++)
            {
               pD->def[i]= pI->pD[i];
               pD->stride[i]= k;
               k*= pD->def[i];
            }
         }
         if (n > 1) { return(4); } // if (pad) WARN();
         else { return( 1 + (0 != pad) ); }
      }
   }
   return(0);
} // checkFD

/***/

template <typename T_Elem>
class CUDAMap
{
protected:
   T_Elem t[BM_NUMT];
   uint     m;

public:
   CUDAMap (const BinMapF32 *pM) { m= pM->m; for (int i=0; i<BM_NUMT; i++) { t[i]= pM->t[i]; } }
   CUDAMap (const BinMapF64 *pM) { m= pM->m; for (int i=0; i<BM_NUMT; i++) { t[i]= pM->t[i]; } }

   __device__ uint eval (const T_Elem f) const
   {
      const uint d= (1 + (f > t[0]) - (f < t[0]));
      return( (m >> d) & 0x1 );
   } // eval
   __device__ uint operator () (const T_Elem f) const { eval(f); }
}; // template class CUDAMap

template <typename T_Elem>
class CUDAFieldMap : protected CUDAMap<T_Elem>
{
protected:
   const T_Elem *pF;

public:
   CUDAFieldMap (const T_Elem * p, const BinMapF32 *pM) : CUDAMap<T_Elem>(pM) { pF= p; }
   CUDAFieldMap (const T_Elem * p, const BinMapF64 *pM) : CUDAMap<T_Elem>(pM) { pF= p; }

   //__device__ uint operator () (const size_t i) const { return CUDAMap<T_Elem>::operator () (pF[i]); }
   __device__ uint operator () (const size_t i) const { return CUDAMap<T_Elem>::eval(pF[i]); }
}; // template class CUDAFieldMap

#if 0

template <typename T_Elem>
class CUDAMultiField
{  // Multiple fields with common stride
protected:
   const T_Elem   * fPtr[BMFI_FIELD_MAX];
   FieldStride stride[3];
   FieldDef    def[3];
   uint        nF;

   void setDS (const FieldDef d[3], const FieldStride *pS)
   {
      if (pS)
      {  // copy
         for (int i=0; i<3; i++) { def[i]= d[i]; stride[i]= pS[i]; }
      }
      else
      {  // Generate stride
         FieldStride k= 1;
         for (int i=0; i<3; i++)
         {
            def[i]= d[i];
            stride[i]= k;
            k*= def[i];
         }
      }
   } // setDS

   uint setF (ConstFieldPtr a[], const uint m)
   {
      uint n= 0;
      for (int i=0; i<BMFI_FIELD_MAX; i++)
      {
         if (m & (0x1 << i)) { fPtr[n]= a[i].p; }
         n+= (NULL != fPtr[i]);
      }
      return(n);
   } // setF

public:
   CUDAMultiField (const BMFieldInfo *pI)
   {
      if (pI->pD)
      {
         setDS(pI->pD, pI->pS);
         nF= setF(pI->field, 0xF);
      }
   } // CTOR

   __device__ T_Elem operator () (const size_t i) const
   {
      T_Elem s= (fPtr[0])[i];
      for (int iF=1; iF < nF; iF++) { s+= (fPtr[iF])[i]; }
      return(s);
   } // operator ()
}; // template class CUDAMultiField

template <typename T_Elem>
class CUDAMultiFieldMap : protected CUDAMultiField<T_Elem>, CUDAMap<T_Elem>
{
public:
   CUDAMultiFieldMap (const BMFieldInfo *pI, const BinMapF32 *pM) : CUDAMultiField<T_Elem>(pI), CUDAMap<T_Elem>(pM) {;}
   CUDAMultiFieldMap (const BMFieldInfo *pI, const BinMapF64 *pM) : CUDAMultiField<T_Elem>(pI), CUDAMap<T_Elem>(pM) {;}

   __device__ uint operator () (const size_t i) const { return CUDAMap<T_Elem>::( CUDAMultiField<T_Elem>::(i) ); }
}; // template class CUDAMultiFieldMap

#endif

/***/

/*
__global__ void vThresh8 (uint r[], const float f[], const size_t n, const BinMapF32 mc)
{
   const size_t i= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ uint z[VT_BLKN];
   if (i < n)
   {
      const int j= i & VT_BLKM;
      const int k= i & 0x7; // j & 0x7
      const int d= 1 + (f[i] > mc.t[0]) - (f[i] < mc.t[0]);
      z[j]= ((mc.m >> d) & 0x1) << k; // smaller shift faster ?

      __syncthreads();

      if (0 == k)
      {  // j : { 0, 8, 16, 24 } 4P, 7I
         for (int l=1; l<8; l++) { z[j]|= z[j+l]; }

         __syncthreads();

         if (0 == j)
         {
            r[i>>VT_BLKS]= ( z[0] << 0 ) | ( z[8] << 8 ) | ( z[16] << 16 ) | ( z[24] << 24 );
         }
      }
   }
} // vThresh8
*/

__device__ int bm1f32 (const float f, const BinMapF32& bm)
{
   const int d= (1 + (f > bm.t[0]) - (f < bm.t[0]));
   return( (bm.m >> d) & 0x1 );
} // bm1f32

__device__ void merge32 (BMPackWord u[32], const int j)
{
/* TODO: consider using CUDA9 warp level primitives...
#define FULL_MASK 0xffffffff
for (int offset = 16; offset > 0; offset /= 2)
    val += __shfl_down_sync(FULL_MASK, val, offset);
*/
   //__syncthreads(); // Unnecessary - no divergence at this point

   if (0 == (j & 0x3))
   {  // j : { 0, 4, 8, 12, 16, 10, 24, 28 } 8P 3I
      for (int l=1; l<4; l++) { u[j]|= u[j+l]; }

      __syncthreads(); // Required for (unexplained) divergence

      if (0 == (j & 0xF))
      {  // j : { 0, 16 } 2P 3I
         for (int l=4; l<16; l+=4) { u[j]|= u[j+l]; }

         __syncthreads(); //  Optional ?
      }
   }
} // merge32

/***/

template <typename T_Elem>
__global__ void mapFieldL32 (BMPackWord r[], const CUDAFieldMap<T_Elem> f, const size_t n)
{
   const size_t i= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ uint z[VT_BLKN];
   if (i < n)
   {
      const int j= i & VT_BLKM;

      z[j]= f(i) << j;

      merge32(z, j);
      if (0 == j) { r[i>>VT_WRDS]= z[0] | z[16]; }
   }
} // mapFieldL32

__global__ void vThreshL32 (BMPackWord r[], const float f[], const size_t n, const BinMapF32 bm)
{
   const size_t i= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ uint z[VT_BLKN];
   if (i < n)
   {
      const int j= i & VT_BLKM;

      z[j]= bm1f32(f[i],bm) << j;

      merge32(z, j);
      if (0 == j) { r[i>>VT_WRDS]= z[0] | z[16]; }
/*
      __syncthreads();

      if (0 == (j & 0x3))
      {  // j : { 0, 4, 8, 12, 16, 10, 24, 28 } 8P 3I
         for (int l=1; l<4; l++) { z[j]|= z[j+l]; }

         __syncthreads();

         //if (0 == j) { r[i>>BLKS]= z[0] | z[4] | z[8] | z[12] | z[16] | z[20] | z[24] | z[28]; }
         if (0 == (j & 0xF))
         {  // j : { 0, 16 } 2P 3I
            for (int l=4; l<16; l+=4) { z[j]|= z[j+l]; }

            __syncthreads();

            if (0 == j) { r[i>>VT_BLKS]= z[0] | z[16]; }
         }
      }
*/
   }
} // vThreshL32

__global__ void vThreshV32
(
   BMPackWord rBM[],
   const CUDAFieldDesc fd,
   const BMOrg bmo,
   const BinMapF32 bm
)
{
   const int x= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ uint u[VT_BLKN];
   if (x < fd.def[0])
   {
      size_t i= x * fd.stride[0] + blockIdx.y * fd.stride[1] + blockIdx.z * fd.stride[2];
      const int j= i & VT_BLKM;

      u[j]= bm1f32( fd.field[0].pF32[i], bm ) << j; // (j & VT_WRDM)

      merge32(u, j);
      if (0 == j) // & VT_WRDM) if BLKS > WRDS !
      {  // (x >> VT_WRDS)
         i= blockIdx.x + blockIdx.y * bmo.rowWS + blockIdx.z * bmo.planeWS;
         rBM[i]= u[0] | u[16];
      }
   }
} // vThreshV32

__global__ void vThreshVSum32
(
   BMPackWord rBM[],
   const CUDAFieldDesc fd,
   const BMOrg bmo,
   const BinMapF32 bm
)
{
   const int x= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ uint u[VT_BLKN];
   if (x < fd.def[0])// && (y < defY))
   {
      size_t i= x * fd.stride[0] + blockIdx.y * fd.stride[1] + blockIdx.z * fd.stride[2];
      const int j= i & VT_BLKM;

      float s= (fd.field[0].pF32)[i];
      for (int f=1; f < fd.nF; f++) { s+= fd.field[f].pF32[i]; }

      u[j]= bm1f32(s,bm) << j;

      merge32(u, j);
      if (0 == j)
      {
         i= (x >> VT_WRDS) + blockIdx.y * bmo.rowWS + blockIdx.z * bmo.planeWS;
         rBM[i]= u[0] | u[16];
      }
   }
} // vThreshVSum32


// DEPRECATED
static int binMapCudaRowsF32
(
   BMPackWord * pBM,
   const F32 * pF,
   const int rowLenF,      // row length ie. "X dimension"
   const int rowStrideBM,  // 32bit word stride of rows of packed binary map, should be >= rowLenF/32
   const int nRows,        // product of other "dimensions" (Y * Z)
   const BinMapF32 *pMC
)
{
   int r= 0;
   CTimerCUDA t;

   if (0 == (rowLenF & VT_BLKM))
   {  // Treat as 1D
      size_t nF= rowLenF * nRows;
      vThreshL32<<<nF/VT_BLKN,VT_BLKN>>>(pBM, pF, nF, *pMC);
      r= (0 == ctuErr(NULL, "vThreshL32()"));
   }
   else
   {  // Hacky 2D - needs proper implementation
      int nBlkRow= (rowLenF + VT_BLKM) / VT_BLKN;
      for (int i=0; i<nRows; i++)
      {
         vThreshL32<<<nBlkRow,VT_BLKN>>>(pBM + i * rowStrideBM, pF + i * rowLenF, rowLenF, *pMC);
      }
      r= (0 == ctuErr(NULL, "nRows*vThreshL32()"));
   }
   LOG("binMapCudaRowsF32(.., L=%d, S=%d, N=%d, BM(%f,0x%X) ) - dt= %Gms\n", rowLenF, rowStrideBM, nRows, pMC->t[0], pMC->m, t.elapsedms());
   //hipDeviceSynchronize(); // stream sync provided by timer
   return(r);
} // binMapCudaRowsF32


/* INTERFACE */

extern "C"
BMOrg *binMapCUDA
(
   BMPackWord  * pW,
   BMOrg       * pO,
   const BMFieldInfo * pF,
   const BinMapF32   * pMC
)
{
   CUDAFieldDesc fd;
   const int id= checkFD(&fd, pF);
   if (id > 0)
   {
      CTimerCUDA t;
      const char * pID= NULL;
      const int   nBlkRow= (fd.def[0] + VT_BLKM) / VT_BLKN;
      setBMO(pO, fd.def, pF->profile);
      if (id <= 2)
      {
#if 0
         binMapCudaRowsF32(pW, fd.field[0].pF32, fd.def[0], pO->rowWS, prodNI(fd.def+1,2), pMC);
         pID= "binMapCudaRowsF32()";
#else
         switch (id)
         {
            case 1 :
            {  const size_t nF= prodNI(fd.def,3);
               mapFieldL32<<<nF/VT_BLKN,VT_BLKN>>>(pW, CUDAFieldMap<float>(fd.field[0].pF32, pMC), nF);
//               vThreshL32<<<nF/VT_BLKN,VT_BLKN>>>(pW, fd.field[0].pF32, nF, *pMC);
               pID= "map()"; // "vThreshL32()";
            }  break;
            case 2 : // Horribly inefficient iteration - only method presently working for !=*32 row length
            {  const int nRows= prodNI(fd.def+1,2);
               for (int i=0; i<nRows; i++)
               {
                  vThreshL32<<<nBlkRow,VT_BLKN>>>(pW + i * pO->rowWS, fd.field[0].pF32 + i * fd.stride[1], fd.def[0], *pMC);
               }
               pID= "nRows*vThreshL32()";
            }  break;
         }
#endif
      }
      else
      {
         const dim3 grd(nBlkRow, fd.def[1], fd.def[2]);
         const dim3 blk(VT_BLKN,1,1);
         switch (id)
         {
            case 3 :
               vThreshV32<<<grd,blk>>>(pW, fd, *pO, *pMC);
               pID= "vThreshV32()";
               break;
            case 4 :
               vThreshVSum32<<<grd,blk>>>(pW, fd, *pO, *pMC);
               pID= "vThreshVSum32()";
               break;
         }
      }
      LOG("binMapCUDA() - %s - dt= %Gms\n", pID, t.elapsedms());
      if (0 == ctuErr(NULL, pID)) { return(pO); }
   }
   return(NULL);
} // binMapCUDA
