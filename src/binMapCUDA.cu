#include "hip/hip_runtime.h"
// binMapCUDA.h - packed binary map generation from scalar fields
// https://github.com/DrAl-HFS/MKF.git
// (c) Project Contributors Jan-June 2019

#include "binMapCUDA.h"


#define BLKS 5
#define BLKD (1<<BLKS)
#define BLKM (BLKD-1)


/***/

__device__ int bm1f32 (const float f, const BinMapF32& bm)
{
   const int d= (1 + (f > bm.t[0]) - (f < bm.t[0]));
   return( (bm.m >> d) & 0x1 );
} // bm1f32

__global__ void vThresh8 (uint r[], const float f[], const size_t n, const BinMapF32 mc)
{
   const size_t i= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ uint z[BLKD];
   if (i < n)
   {
      const int j= i & BLKM;
      const int k= i & 0x7; // j & 0x7
      const int d= 1 + (f[i] > mc.t[0]) - (f[i] < mc.t[0]);
      z[j]= ((mc.m >> d) & 0x1) << k; // smaller shift faster ?

      __syncthreads();

      if (0 == k)
      {  // j : { 0, 8, 16, 24 } 4P, 7I
         for (int l=1; l<8; l++) { z[j]|= z[j+l]; }

         __syncthreads();

         if (0 == j)
         {
            r[i>>BLKS]= ( z[0] << 0 ) | ( z[8] << 8 ) | ( z[16] << 16 ) | ( z[24] << 24 );
         }
      }
   }
} // vThresh8
/*
__global__ void vThresh32 (uint r[], const float f[], const size_t n, const BinMapF32 mc)
{
   const size_t i= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ uint z[BLKD];
   if (i < n)
   {
      const int j= i & BLKM;
      const int d= 1 + (f[i] > mc.t[0]) - (f[i] < mc.t[0]);
      z[j]= ((mc.m >> d) & 0x1) << j; // assume "barrel" shifter

      __syncthreads();

      if (0 == (j & 0x3))
      {  // j : { 0, 4, 8, 12, 16, 10, 24, 28 } 8P 3I
         for (int l=1; l<4; l++) { z[j]|= z[j+l]; }

         __syncthreads();

         //if (0 == j) { r[i>>BLKS]= z[0] | z[4] | z[8] | z[12] | z[16] | z[20] | z[24] | z[28]; }
         if (0 == (j & 0xF))
         {  // j : { 0, 16 } 2P 3I
            for (int l=4; l<16; l+=4) { z[j]|= z[j+l]; }

            __syncthreads();

            if (0 == j) { r[i>>BLKS]= z[0] | z[16]; }
         }
      }
   }
} // vThresh32
*/
