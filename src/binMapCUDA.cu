#include "hip/hip_runtime.h"
// binMapCUDA.h - packed binary map generation from scalar fields
// https://github.com/DrAl-HFS/MKF.git
// (c) Project Contributors Jan-Sept 2019

#include "binMapCUDA.h"
#include "utilCUDA.hpp"


#define VT_WRDS 5
#define VT_WRDN (1<<VT_WRDS)
#define VT_WRDM (VT_WRDN-1)

#define VT_BLKS 7             // 7
#define VT_BLKN (1<<VT_BLKS)
#define VT_BLKM (VT_BLKN-1)

#define VT_BWS (VT_BLKS - VT_WRDS)

#define CAT_NUM 2

/***/

bool validPtr (const void *p) { return(NULL != p); }

uint copyValidPtrByMask (ConstFieldPtr r[], const int max, const ConstFieldPtr a[], const uint mask)
{
   uint t, i= 0, n= 0;
   do
   {
      t= (0x1 << i);
      if (validPtr(a[i].p) && (mask & t))
      {
         r[n++]= a[i];
      }
      i++;
   } while ((mask > t) && (n < max));
   //if (n < max) { r[n].p= NULL; } guard ?
   return(n);
} // copyValidPtrByMask

uint countValidPtrByMask (const ConstFieldPtr a[], uint mask)
{
   uint i= 0, n= 0;
   do
   {
      n+= validPtr(a[i].p) && (mask & 0x1);
      mask >>= 1;
      i++;
   } while (mask > 0);
   return(n);
} // copyValidPtrByMask

struct Region
{  // Expect multiple fields, common def & stride
   size_t   nElem;
   FieldDef elemDef[3];
   int      grdDef0;
   uint16_t blkDef0;
   uint8_t  nD, nF;

//public:
   bool validate (const BMFieldInfo *pI) // TODO: SubRegion/Box ???
   {
      if (pI)
      {
         nD= 0; nF= 0;
         if (pI->pFieldDevPtrTable)
         {
            nF= countValidPtrByMask(pI->pFieldDevPtrTable, pI->fieldTableMask);
         }
         if (pI->pD)
         {
            for (int i=0; i<3; i++)
            {
               elemDef[nD]= pI->pD[i];
               nD+= (elemDef[nD] > 1);
            }
            for (int i=nD; i<3; i++) { elemDef[i]= 1; }
            blkDef0= VT_BLKN;
            //if (blkDim0...
            grdDef0= (elemDef[0] + (blkDef0-1) ) / blkDef0;
            nElem= prodNI(elemDef,3);
         }
         return((nD > 0) && (nF > 0));
      }
      return(false);
   } // validate (as conditional 'CTOR')

   bool collapsable (void) const { return( (elemDef[0] == nElem) || (0 == (elemDef[0] & VT_WRDM)) ); }

   dim3 blockDef (void) { return dim3(blkDef0, 1, 1); }
   dim3 gridDef (void) { return dim3(grdDef0, elemDef[1], elemDef[2]); }
   int collapsedBlockDef (void) { return(blkDef0); }
   int collapsedGridDef (void) { if (blkDef0 > 0) { return((nElem + blkDef0-1) / blkDef0); } else return(0); }
}; // struct Region


/* Templated device classes */

template <typename T_Elem>
class CUDAImgMom
{
protected:
   T_Elem * pM2;

   CUDAImgMom (void *p) { pM2= static_cast<T_Elem*>(p); }

   __device__ T_Elem sum (const T_Elem v) // const ? // Block-wide sum reduction
   {
      __shared__ T_Elem t[VT_BLKN];
      t[threadIdx.x]= v;
      for (int s= blockSize.x>>1; s > 0; s>>= 1)
      {  __syncthreads(); // Keep block together
         if (threadIdx.x < s) { t[threadIdx.x]+= t[threadIdx.x+s]; }
      }

      return(t[threadIdx.x]);
   } // sum
/*
   __device__ void sumCat (T_Elem *pR, const int strideR, const T_Elem v, const int c) // Block-wide categorical sum reduction
   {
      __shared__ T_Elem t[CAT_NUM][VT_BLKN];
#if CAT_NUM > 2
      for (int i=0; i<CAT_NUM; i++) { t[i][threadIdx.x]= 0; }
#else
      t[c^0x1][threadIdx.x]= 0;
#endif
      t[c][threadIdx.x]= v;
      for (int s= blockSize.x>>1; s > 0; s>>= 1)
      {  __syncthreads(); // Keep block together
         if (threadIdx.x < s) { t[c][threadIdx.x]+= t[c][threadIdx.x+s]; }
      }
      // NOT SUFFICIENT! Only t[c][0] will be correct, other categories incomplete partial in t[x][y]
      if (0 == threadIdx.x)
      {
         for (int i=0; i<CAT_NUM; i++) { atomicAdd( pR+i*strideR, t[i][0] ); }
      }
      __syncthreads();
   } // sumCat
*/
   __device__ void add (int x, int y, int z, T_Elem m)
   {
      T_Elem s, m2= m * m;

      s= sum(m);
      if (0 == threadIdx.x) { atomicAdd( pM2+0, s); }
      s= sum(m2);
      if (0 == threadIdx.x) { atomicAdd( pM2+1, s); }
   }
}; // class CUDAImgMom

template <typename T_Elem>
class CUDAMap
{
protected:
   T_Elem   t[BM_NUMT];
   uint     m;

public:
   CUDAMap (const BinMapF32 *pM) { m= pM->m; for (int i=0; i<BM_NUMT; i++) { t[i]= pM->t[i]; } }
   CUDAMap (const BinMapF64 *pM) { m= pM->m; for (int i=0; i<BM_NUMT; i++) { t[i]= pM->t[i]; } }

   __device__ uint eval (const T_Elem f) const
   {
      const uint d= (1 + (f > t[0]) - (f < t[0]));
      return( (m >> d) & 0x1 );
   } // eval
   __device__ uint operator () (const T_Elem f) const { eval(f); }
}; // template class CUDAMap

template <typename T_Elem>
class CUDAFieldMap : protected CUDAMap<T_Elem>
{
protected:
   const T_Elem * pF;

public:                 // static_cast< const T_Elem * >() - irrelevant so why bother?
   CUDAFieldMap (const BMFieldInfo *pI, const BinMapF32 *pM) : CUDAMap<T_Elem>(pM)
   { copyValidPtrByMask( (ConstFieldPtr*)&pF, 1, pI->pFieldDevPtrTable, pI->fieldTableMask); }
   CUDAFieldMap (const BMFieldInfo *pI, const BinMapF64 *pM) : CUDAMap<T_Elem>(pM)
   { copyValidPtrByMask( (ConstFieldPtr*)&pF, 1, pI->pFieldDevPtrTable, pI->fieldTableMask); }

   //__device__ uint operator () (const size_t i) const { return CUDAMap<T_Elem>::operator () (pF[i]); }
   __device__ uint operator () (const size_t i) const { return CUDAMap<T_Elem>::eval(pF[i]); }
}; // template class CUDAFieldMap

class CUDAOrg
{
   FieldStride fs[3];
   uint  rowElem;
   BMStride rowWS, planeWS;   // 32b word strides

   void setDS (const FieldStride *pS, const FieldDef *pD)
   {
      if (pS)
      {  // copy
         for (int i=0; i<3; i++) { fs[i]= pS[i]; }
      }
      else if (pD)
      {  // Generate stride
         FieldStride k= 1;
         for (int i=0; i<3; i++)
         {
            fs[i]= k;
            k*= pD[i];
         }
      }
   } // setDS

public:

   CUDAOrg (const BMOrg *pO, const BMFieldInfo *pI)
   {
      setDS(pI->pS, pI->pD);
      rowElem= pO->rowElem;
      rowWS=   pO->rowWS;
      planeWS= pO->planeWS;
   } // CTOR

   __device__ bool inRow (uint x) const { return(x < rowElem); }
   __device__ size_t fieldIndex (uint x, uint y, uint z) const { return(x * fs[0] + y * fs[1] + z * fs[2]); }
   __device__ size_t bmIndex (uint x, uint y, uint z) const { return((x >> VT_WRDS) + y * rowWS + z * planeWS); }

}; // CUDAOrg


template <typename T_Elem>
class CUDAMultiField
{  // Multiple fields with common stride
protected:
   const T_Elem   * fPtrTab[BMFI_FIELD_MAX];
   uint           nF;

public:
   CUDAMultiField (const BMFieldInfo *pI)
   {
      if (pI->pD)
      {
         nF= copyValidPtrByMask( (ConstFieldPtr*)fPtrTab, BMFI_FIELD_MAX, pI->pFieldDevPtrTable, pI->fieldTableMask);
      }
   } // CTOR

   //__device__ size_t index (uint x, uint y, uint z) const { return(x * stride[0] + y * stride[1] + z * stride[2]); }

   __device__ T_Elem sum (const size_t i) const
   {
      T_Elem s= (fPtrTab[0])[i];
      for (int iF=1; iF < nF; iF++) { s+= (fPtrTab[iF])[i]; }
      return(s);
   } // sum
   __device__ T_Elem operator () (const size_t i) const { return sum(i); }
}; // template class CUDAMultiField

// TODO: polymorphisation of CUDAFieldMap & CUDAMultiFieldMap ... ?
template <typename T_Elem>
class CUDAMultiFieldMap : protected CUDAMultiField<T_Elem>, CUDAMap<T_Elem>
{
public:
   CUDAMultiFieldMap (const BMFieldInfo *pI, const BinMapF32 *pM) : CUDAMultiField<T_Elem>(pI), CUDAMap<T_Elem>(pM) {;}
   CUDAMultiFieldMap (const BMFieldInfo *pI, const BinMapF64 *pM) : CUDAMultiField<T_Elem>(pI), CUDAMap<T_Elem>(pM) {;}

   //__device__ size_t index (uint x, uint y, uint z) const { return CUDAMultiField<T_Elem>::index(x,y,z); }
   __device__ uint operator () (const size_t i) const { return CUDAMap<T_Elem>::eval( CUDAMultiField<T_Elem>::sum(i) ); }
}; // template class CUDAMultiFieldMap


/* Device utility functions */
//#define NO_WLP
//#define MERGE32_SAFE

#ifdef NO_WLP

__device__ uint mergeOR32 (volatile uint w[32], const int lane)
{
#ifdef MERGE32_SAFE
   for (int s= 16; s > 0; s>>= 1)
   {  __syncthreads(); // Slightly slower but keeps warp together
      if (lane < s) { w[lane]|= w[lane+s]; }
   }
#else
   if (lane < 16) // DANGER! Half warp will free-run to next sync -
   {  // DO NOT USE when warp-wide functions are subsequently employed!
      #pragma unroll 5 // Ineffective? Presume limited by shared read/write
      for (int s= 16; s > 0; s>>= 1) { __syncthreads(); w[lane]|= w[lane+s]; }
   }
#endif
   return(w[lane]);
} // mergeOR32

__device__ uint bitMergeShared (uint v)
{
   __shared__ BMPackWord w[VT_BLKN]; // Caution! fixed blocksize!

   w[threadIdx.x]= v;
   return mergeOR32(w + (threadIdx.x & ~VT_WRDM), threadIdx.x & VT_WRDM);
} // bitMergeShared

#define BIT_MERGE(w) bitMergeShared(w)

#else // NO_WLP

__device__ uint bitMergeBallot (uint v)
{  // Nice simplification but doesn't work... ?
   return __ballot_sync(VT_WRDM, v);
} // bitMergeBallot

#define SHFL_MASK_ALL (-1) // (1<<warpSize)-1 or 0xFFFFFFFF

// TODO - FIX - warning: integer conversion resulted in a change of sign
// #pragma NVCC warning disable ?

__device__ uint bitMergeWarp (uint w)
{  // CUDA9 warp level primitives (supported on CUDA7+ ? )
#if 1
   #pragma unroll 5 // Seems to have no effect - need to enable a setting ?
   for (int s= warpSize/2; s > 0; s>>= 1) { w|= __shfl_down_sync(SHFL_MASK_ALL, w, s); }
#else // Manual unroll is faster (but reducing mask in successive steps makes negligible difference)
   w|= __shfl_down_sync(SHFL_MASK_ALL, w, 16);
   w|= __shfl_down_sync(SHFL_MASK_ALL, w, 8);
   w|= __shfl_down_sync(SHFL_MASK_ALL, w, 4);
   w|= __shfl_down_sync(SHFL_MASK_ALL, w, 2);
   w|= __shfl_down_sync(SHFL_MASK_ALL, w, 1);
#endif
   return(w);
} // bitMergeWarp

#define BIT_MERGE(w) bitMergeWarp(w)

#endif // NO_WLP

/* KERNELS */

template <typename T_Elem>
__global__ void mapField (BMPackWord rBM[], const CUDAFieldMap<T_Elem> f, const size_t n)
{
   const size_t i= blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n)
   {
      const uint j= threadIdx.x & VT_WRDM;   // lane index (bit number)

      uint w= BIT_MERGE( f(i) << j );

      if (0 == j) { rBM[i>>VT_WRDS]= w; }
   }
} // mapField

template <typename T_Elem>
__global__ void mapMultiField (BMPackWord rBM[], const CUDAMultiFieldMap<T_Elem> mf, const size_t n)
{
   const size_t i= blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n)
   {
      const uint j= threadIdx.x & VT_WRDM;   // lane index (bit number)

      uint w= BIT_MERGE( mf(i) << j );

      if (0 == j) { rBM[i>>VT_WRDS]= w; }
   }
} // mapMultiField

template <typename T_Elem>
__global__ void mapStrideField (BMPackWord rBM[], const CUDAOrg org, const CUDAFieldMap<T_Elem> f)
{
   const uint x= blockIdx.x * blockDim.x + threadIdx.x;
   if (org.inRow(x))
   {
      size_t i= org.fieldIndex(x, blockIdx.y, blockIdx.z);
      const uint j= threadIdx.x & VT_WRDM;   // lane index (bit number)

      uint w= BIT_MERGE( f(i) << j );

      if (0 == j) { rBM[org.bmIndex(x, blockIdx.y, blockIdx.z)]= w; }
   }
} // mapStrideField

template <typename T_Elem>
__global__ void mapStrideMultiField (BMPackWord rBM[], const CUDAOrg org, const CUDAMultiFieldMap<T_Elem> mf) // const size_t n)
{
   const uint x= blockIdx.x * blockDim.x + threadIdx.x;
   if (org.inRow(x))
   {
      size_t i= org.fieldIndex(x, blockIdx.y, blockIdx.z);
      const uint j= threadIdx.x & VT_WRDM;

      uint w= BIT_MERGE( mf(i) << j );

      if (0 == j) { rBM[org.bmIndex(x, blockIdx.y, blockIdx.z)]= w; }
   }
} // mapStrideMultiField


/* INTERFACE */

extern "C"
BMOrg *binMapCUDA
(
   BMPackWord  * pW,
   BMOrg       * pO,
   const BMFieldInfo * pF,
   const BinMapF32   * pMC
)
{
   Region reg;

   //if (32 != warpSize) { WARN("[binMapCUDA] warpSize=%d\n", warpSize); }
   if (reg.validate(pF))
   {
      CTimerCUDA t;
      const char * pID= NULL;

      if ( setBMO(pO, reg.elemDef, pF->profID) )
      {
         //LOG("Region::validate() - D%d F%d\n", reg.nD, reg.nF);
         switch (pF->profID & 0x30)
         {
            case 0x00 :
            if (reg.collapsable() && (1 == reg.nF))
            {
               mapField<<< reg.collapsedGridDef(), reg.collapsedBlockDef() >>>(pW, CUDAFieldMap<float>(pF, pMC), reg.nElem);
               pID= "mapField()";
               break;
            } // else...
            case 0x10 :
            if (1 == reg.nF)
            {
               mapStrideField<<< reg.gridDef(), reg.blockDef() >>>(pW, CUDAOrg(pO, pF), CUDAFieldMap<float>(pF, pMC));
               pID= "mapStrideField()";
               break;
            } // else...
            case 0x20 :
            if (reg.collapsable())
            {
               mapMultiField<<< reg.collapsedGridDef(), reg.collapsedBlockDef() >>>(pW, CUDAMultiFieldMap<float>(pF, pMC), reg.nElem);
               pID= "mapMultiField()";
               break;
            } // else ...
            case 0x30 :
               mapStrideMultiField<<< reg.gridDef(), reg.blockDef() >>>(pW, CUDAOrg(pO, pF), CUDAMultiFieldMap<float>(pF, pMC));
               pID= "mapStrideMultiField()";
               break;
            /*case never :
            {  const int nRows= prodNI(fd.def+1,2);
               for (int i=0; i<nRows; i++)
               {  // Horribly inefficient iteration
                  mapField<<<reg.nBlk,VT_BLKN>>>(pW + i * pO->rowWS, CUDAFieldMap<float>(fd.field[0].pF32 + i * fd.stride[1], pMC), fd.def[0]);
               }
               pID= "nRows*mapField()"; break;
            }*/
         }
         LOG("binMapCUDA() - %s<<<%u>>>() - dt= %Gms\n", pID, reg.collapsedBlockDef(), t.elapsedms());
         if (0 == ctuErr(NULL, pID)) { return(pO); }
      }
   }
   return(NULL);
} // binMapCUDA
