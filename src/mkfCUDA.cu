#include "hip/hip_runtime.h"
// mkf.cu - Minkowski Functional pattern processing using CUDA NB: .cu assumes c++ style compilation
// https://github.com/DrAl-HFS/MKF.git
// (c) Project Contributors Jan-June 2019

#ifndef MKF_CUDA_CU
#define MKF_CUDA_CU // supress header "multiple definition" glitch
#endif

#include "mkfCUDA.h"

#ifdef MKF_CUDA_CU
#undef MKF_CUDA_CU // header glitch supression done
#endif

// Wide counter for atomicAdd (nvcc dislikes size_t)
typedef unsigned long long CUACount;


/***/

// CUDA kernels and wrappers

#define BLKS 5
#define BLKD (1<<BLKS)
#define BLKM (BLKD-1)
#define BINS MKF_BINS
#define BINM (BINS-1)
#define BINS16P (BINS/2)
//define BLKN 1024/BLKD

__global__ void vThresh8 (uint r[], const float f[], const size_t n, const BinMapF32 mc)
{
   const size_t i= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ uint z[BLKD];
   if (i < n)
   {
      const int j= i & BLKM;
      const int k= i & 0x7; // j & 0x7
      const int d= 1 + (f[i] > mc.t[0]) - (f[i] < mc.t[0]);
      z[j]= ((mc.m >> d) & 0x1) << k; // smaller shift faster ?

      __syncthreads();

      if (0 == k)
      {  // j : { 0, 8, 16, 24 } 4P, 7I
         for (int l=1; l<8; l++) { z[j]|= z[j+l]; }

         __syncthreads();

         if (0 == j)
         {
            r[i>>BLKS]= ( z[0] << 0 ) | ( z[8] << 8 ) | ( z[16] << 16 ) | ( z[24] << 24 );
         }
      }
   }
} // vThresh8

__global__ void vThresh32 (uint r[], const float f[], const size_t n, const BinMapF32 mc)
{
   const size_t i= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ uint z[BLKD];
   if (i < n)
   {
      const int j= i & BLKM;
      const int d= 1 + (f[i] > mc.t[0]) - (f[i] < mc.t[0]);
      z[j]= ((mc.m >> d) & 0x1) << j; // assume "barrel" shifter

      __syncthreads();

      if (0 == (j & 0x3))
      {  // j : { 0, 4, 8, 12, 16, 10, 24, 28 } 8P 3I
         for (int l=1; l<4; l++) { z[j]|= z[j+l]; }

         __syncthreads();

         //if (0 == j) { r[i>>BLKS]= z[0] | z[4] | z[8] | z[12] | z[16] | z[20] | z[24] | z[28]; }
         if (0 == (j & 0xF))
         {  // j : { 0, 16 } 2P 3I
            for (int l=4; l<16; l+=4) { z[j]|= z[j+l]; }

            __syncthreads();

            if (0 == j) { r[i>>BLKS]= z[0] | z[16]; }
         }
      }
   }
} // vThresh32

#define CHUNK_SHIFT (5)
#define CHUNK_SIZE (1<<CHUNK_SHIFT)
#define CHUNK_MASK (CHUNK_SIZE-1)

typedef unsigned long long ULL;
typedef uint   U16P;

//uint froodldoodl=0;

class ChunkBuf
{
   ULL u00, u01, u10, u11;

   __device__ uint buildNext (void)
   {
      uint bp=  ( u00 & 0x3);       u00 >>= 1;
      bp|= ((u01 & 0x3) << 2); u01 >>= 1;
      bp|= ((u10 & 0x3) << 4); u10 >>= 1;
      bp|= ((u11 & 0x3) << 6); u11 >>= 1;
      return(bp);
   }

public:
   __device__ ChunkBuf (const uint * __restrict__ pR0, const uint * __restrict__ pR1, const int rowStride)
   {
      u00= pR0[0];
      u01= pR0[rowStride];
      u10= pR1[0];
      u11= pR1[rowStride];
   }
   __device__ void loadSh1 (const uint * __restrict__ pR0, const uint * __restrict__ pR1, const int rowStride)
   {
      u00|= ( (ULL) pR0[0] ) << 1;
      u01|= ( (ULL) pR0[rowStride] ) << 1;
      u10|= ( (ULL) pR1[0] ) << 1;
      u11|= ( (ULL) pR1[rowStride] ) << 1;
   }
   __device__ void add (uint bpfd[BINS], const int n)
   {
      for (int i= 0; i < n; i++)
      {
         uint bp= buildNext();
         bpfd[ bp ]++;
      }
   } // add
   __device__ void addU16P (U16P bpfd[BINS16P], const int n)
   {
      const uint w[2]={1,1<<16}; // even -> lo, odd -> hi (16b)

      for (int i= 0; i < n; i++)
      {
         uint bp= buildNext();
         bpfd[ bp >> 1 ]+= w[bp & 1];
      }
   } // addU16P
}; // class ChunkBuf

__device__ void addRowBPFD
(
   uint         bpfd[BINS], // result pattern distribution
   const uint  * __restrict__ pRow[2],
   const int   rowStride,
   const int   n    // Number of single bit elements packed in row
)
{  // seq
   int m, k, i;
   ChunkBuf  cb(pRow[0]+0, pRow[1]+0, rowStride);
   k= MIN(CHUNK_SIZE-1, n-1); //dbg[0]+= k;
   cb.add(bpfd, k);
   // Subsequent whole chunks yield n patterns
   i= 0;
   m= n>>CHUNK_SHIFT;
   while (++i < m)
   {
      cb.loadSh1(pRow[0]+i, pRow[1]+i, rowStride);
      cb.add(bpfd, CHUNK_SIZE);
   }
   // Check for residual bits < CHUNK_SIZE
   k= n & CHUNK_MASK;
   if (k > 0)
   {
      cb.loadSh1(pRow[0]+i, pRow[1]+i, rowStride); // ensure LSB aligned?
      cb.add(bpfd, k); //dbg[2]+= k;
   }
} // addRowBPFD

__device__ void zeroBins (uint bpfd[BINS*BLKD], const int row, const int bins)
{
   for (int k= row; k < BINS; k+= blockDim.x)
   {  // (transposed zeroing for write coalescing)
      for (int j= 0; j < blockDim.x; j++) { bpfd[j*BINS+k]= 0; }
   }
} // zeroBins

__device__ void reduceBins (CUACount rBPFD[BINS], const uint bpfd[BINS*BLKD], const int row, const int bins)
{
   for (int k= row; k < bins; k+= blockDim.x)
   {  // (transposed reduction for read coalescing)
      CUACount t= 0;
      for (int j= 0; j < blockDim.x; j++) { t+= bpfd[j*bins+k]; }
      atomicAdd( rBPFD+k, t );
   }
} // reduceBins

//BINS16P
__device__ void reduceBinsU16P (CUACount rBPFD[BINS], const U16P bpfd[BINS16P*BLKD], const int row, const int bins)
{
   for (int k= row; k < bins; k+= blockDim.x)
   {  // (transposed reduction for read coalescing)
      CUACount t[2]= {0,0};
      for (int j= 0; j < blockDim.x; j++)
      {
         const U16P u= bpfd[j*bins+k];
         t[0]+= u & 0xFFFF;
         t[1]+= u >> 16;
      }
      const int i= k<<1;
      atomicAdd( rBPFD+i, t[0] );
      atomicAdd( rBPFD+i+1, t[1] );
   }
} // reduceBins

__global__ void addPlaneBPFD (CUACount rBPFD[BINS], const uint * pPln0, const uint * pPln1, const int rowStride, const int defW, const int defH)
{
   const size_t i= blockIdx.x * blockDim.x + threadIdx.x; // ???
   const int row= i & BLKM;
   __shared__ uint bpfd[BINS*BLKD]; // 32KB

   //if (blockDim.x > BLKD) { printf("ERROR: addPlaneBPFD() - blockDim=%d", blockDim.x); return; }
   //else { printf(" - blockDim=%d,%d,%d\n", blockDim.x, blockDim.y, blockDim.z); }
   zeroBins(bpfd, row, BINS);
   if (i < defH)
   {
      const U32 * pRow[2]= { pPln0 + i*rowStride, pPln1 + i*rowStride };

      addRowBPFD(bpfd+row*BINS, pRow, rowStride, defW);
   }
   __syncthreads();
   reduceBins(rBPFD, bpfd, row, BINS);
} // addPlaneBPFD


/***/

hipError_t ctuErr (hipError_t *pE, const char *s)
{
   hipError_t e;
   if (NULL == pE) { e= hipGetLastError(); } else { e= *pE; }
   if (0 != e)
   {
      ERROR("%s - r=%d -> %s\n", s, e, hipGetErrorName(e));
   }
   return(e);
} // ctuErr

extern "C" int mkfCUDAGetBPFDSimple (Context *pC, const int def[3], const BinMapF32 *pBM)
{
   hipError_t r;
   int blkD= BLKD;//256;
   int nBlk= 0;

   if (pC->pHF)
   {
      //r= hipMemcpy(pC->pDU, pC->pHU, pC->bytesU, hipMemcpyHostToDevice);

      if (NULL == pC->pDF)
      {
         r= hipMalloc(&(pC->pDF), pC->bytesF);
         ctuErr(&r, "hipMalloc()");
      }
      if (pC->pDF)
      {
         r= hipMemcpy(pC->pDF, pC->pHF, pC->bytesF, hipMemcpyHostToDevice);
         ctuErr(&r, "hipMemcpy()");
      }

      if (NULL == pC->pDU)
      {
         r= hipMalloc(&(pC->pDU), pC->bytesU);
         ctuErr(&r, "hipMalloc()");
      }

      if (pC->pDF && pC->pDU)
      {
         if (pC->pDZ) { hipMemset(pC->pDZ, 0, pC->bytesZ); }
         if (pC->nF <= blkD) { blkD= BLKD; }
         nBlk= (pC->nF + blkD-1) / blkD;
         LOG("***\nmkfCUDAGetBPFDSimple() - bmc: %f,0x%X\n",pBM->t[0], pBM->m);
         // CAVEAT! Treated as 1D
         vThresh32<<<nBlk,blkD>>>(pC->pDU, pC->pDF, pC->nF, *pBM);
         ctuErr(NULL, "vThresh32()");
         hipDeviceSynchronize();

         if (pC->pHU)
         {
            LOG("hipMemcpy(%p, %p, %u)\n", pC->pHU, pC->pDU, pC->bytesU);
            r= hipMemcpy(pC->pHU, pC->pDU, pC->bytesU, hipMemcpyDeviceToHost);
            ctuErr(NULL, "{vThresh32+} hipMemcpy()");
         }
      }
   }
   else if (pC->pHU)
   {
      r= hipMemcpy(pC->pDU, pC->pHU, pC->bytesU, hipMemcpyHostToDevice);
      if (NULL == pC->pDZ)
      {
         r= hipMalloc(&(pC->pDZ), pC->bytesZ);
         ctuErr(&r, "hipMalloc()");
      }
      if (pC->pDZ) { hipMemset(pC->pDZ, 0, pC->bytesZ); }
   }
   if (pC->pDU && pC->pDZ)
   {
      //size_t bpdBytes= 256*sizeof(uint);
      //if ((pC->pDZ) && (pC->bytesZ >= bpdBytes))
      CUACount *pBPFD= (CUACount*)(pC->pDZ);
      const int rowStride= def[0] / 32;
      const int nRowPairs= def[1]-1;
      const int nPlanePairs= def[2]-1;
      const int planeStride= def[1] * rowStride;

      //if (nRowPairs <= blkD) {
      blkD= BLKD;
      nBlk= (nRowPairs + blkD-1) / blkD;

      for (int i= 0; i < nPlanePairs; i++)
      {
         const U32 *pP0= pC->pDU + i * planeStride;
         const U32 *pP1= pC->pDU + (i+1) * planeStride;
         addPlaneBPFD<<<nBlk,blkD>>>(pBPFD, pP0, pP1, rowStride, def[0], nRowPairs); //, pBPFD+256);
         if (0 != ctuErr(NULL, "addPlane"))
         { LOG(" .. <<<%d,%d>>>(%p, %p, %p ..)\n", nBlk, blkD, pBPFD, pP0, pP1); }
      }
      hipDeviceSynchronize();
      if (pC->pHZ)
      {
         r= hipMemcpy(pC->pHZ, pC->pDZ, pC->bytesZ, hipMemcpyDeviceToHost);
         ctuErr(&r, "{addPlane+} hipMemcpy()");
      }
   }
   return(1);
} // mkfCUDAGetBPFDSimple


#ifdef MKF_CUDA_MAIN

#if 0
__global__ void vAddB (float r[], const float a[], const float b[], const int n)
{
   int i= blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) { r[i]= a[i] + b[i]; }
} // vAddB

void sanityTest (Context *pC)
{
   const int n= 1024;
   int i, e=0;
   for (i=0; i<n; i++) { pC->pHF[i]= i; pC->pHF[2*n - (1+i)]= 1+i; }
   hipMemcpy(pC->pDF, pC->pHF, 2*n*sizeof(pC->pHF[0]), hipMemcpyHostToDevice); ctuErr(NULL, "hipMemcpy 1");
   vAddB<<<8,128>>>(pC->pDF+2*n, pC->pDF+0, pC->pDF+n, n);
   hipMemcpy(pC->pHF+2*n, pC->pDF+2*n, n*sizeof(pC->pHF[0]), hipMemcpyDeviceToHost); ctuErr(NULL, "hipMemcpy 2");

   i= 2 * n;
   LOG("sanityTest() - vAddB() - [%d]=%G", i, pC->pHF[i]);
   for ( ; i < (3*n)-1; i++)
   {
      if (pC->pHF[i] != n) { ++e; LOG(" [%d]=%G", i, pC->pHF[i]); }
   }
   LOG(" [%d]=%G\n", i, pC->pHF[i]);

   printf("*e=%d*\n", e);
} // sanityTest();
#endif

#include "geomHacks.h"
#include "mkfUtil.h"

int buffAlloc (Context *pC, const int def[3], const int blkZ)
{
   int vol= def[0] * def[1] * def[2];

   pC->nF= vol;
   pC->bytesF= sizeof(*(pC->pHF)) * pC->nF;
   pC->nU= BITS_TO_WRDSH(vol,5);
   pC->bytesU= sizeof(*(pC->pHU)) * pC->nU;
   pC->nZ= BLKD + blkZ * 256;
   pC->bytesZ= 8 * pC->nZ; // void * sizeof(*(pC->pHZ))

   LOG("F: %zu -> %zu Bytes\nU: %zu -> %zu Bytes\n", pC->nF, pC->bytesF, pC->nU, pC->bytesU);

   return cuBuffAlloc(pC,vol);
} // buffAlloc


static const char gSepCh[2]={' ','\n'};

void dumpF (const float f[], const int n, const int wrap)
{
   int i=0;
   while (i<n)
   {
      int k= i + wrap;
      if (k > n) { k= n; }
      for (int j= i; j < k; j++) { LOG("%G%c", f[j], gSepCh[(j+1)==k]); }
      i= k;
   }
} // dumpF

void dumpUX (const uint u[], const int n, const int wrap)
{
   int i=0;
   while (i<n)
   {
      int k= i + wrap;
      if (k > n) { k= n; }
      for (int j= i; j < k; j++) { LOG("%08X%c", u[j], gSepCh[(j+1)==k]); }
      i= k;
   }
} // dumpUX

size_t bitCountNU32 (U32 u[], const int n)
{
   size_t t= 0;
   for (int i= 0; i<n; i++) { t+= BIT_COUNT_Z(u[i]); }
   return(t);
} // bitCountNU32

void mkft (Context *pC, const int def[3])
{
   BinMapF32 bmc;

   //dumpF(pC->pHF+n, n, def[0]);
   setBinMapF32(&bmc,">=",0.5);
   mkfCUDAGetBPFDSimple(pC, def, &bmc);
#if 0
   size_t t= bitCountNU32(pC->pHU, pC->bytesU>>2);
   LOG("bitCountNU32() -> %zu\n", t);
   LOG("%p[%u]:\n",pC->pHU,pC->nU);
   m= def[0] >> BLKS; // def[0] / BLKD;
   n= m * def[1];
   if (n > pC->nU/2) { n= pC->nU/2; }
   while ((m<<1) < 16) { m<<= 1; }
   dumpUX(pC->pHU+2*n, n, m);
   LOG("%s\n","-");
   dumpUX(pC->pHU+3*n, n, m);
#endif
   if (pC->pHZ)
   {
      const size_t *pBPFD= (size_t*)pC->pHZ;
      size_t s= 0;
      LOG("\tvolFrac=%G chiEP=%G\n", volFrac(pBPFD), chiEP3(pBPFD));

      for (int i= 0; i<256; i++)
      {
         if (pBPFD[i] > 0) { LOG("[0x%X]=%u\n", i, pBPFD[i]); }
      }
      for (int r= 0; r<BLKD; r++)
      {
         s+= pBPFD[256+r];
         //LOG("r%d=%u\n", r, pBPFD[256+r]);
      }
      LOG("s=%zu\n", s);
   }
} // mkft

int main (int argc, char *argv[])
{
   const int def[3]= {64,64,2};
   Context cux={0};

   if (buffAlloc(&cux, def, 1))
   {
      //sanityTest(&cux);
      if (0)
      {
         genPattern(cux.pHF, 1, def, 0.5*def[1] - 0.5);
         mkft(&cux,def);
      }
      else
      {
         Context t= cux;
         const int wDef= def[0] >> 5;
         const int lDef= def[1] * def[2];
         t.pHF= t.pDF= NULL;

         for (int i= 0; i < lDef; i++)
         {
            int j= wDef * i;
            t.pHU[j+0]= 0x7FFFFFFF; // NB: L to R order -> bits 0 to 31
            t.pHU[j+1]= 0xFFFFFFFF;
            //t.pHU[j+2]= 0xFFFFFFFF;
            //t.pHU[j+3]= 0xFFFFFFFF;
         }
         mkft(&t,def);
         for (int i= 0; i < lDef; i++)
         {
            int j= wDef * i;
            t.pHU[j+0]= 0xFFFFFFFF;
            t.pHU[j+1]= 0xFFFFFFFE;
            //t.pHU[j+2]= 0xFFFFFFFF;
         }
         mkft(&t,def);
      }
      cuBuffRelease(&cux);
   }
   hipDeviceReset();
} // main

#endif // MKF_CUDA_MAIN
