#include "hip/hip_runtime.h"
// mkf.cu - Minkowski Functional pattern processing using CUDA NB: .cu assumes c++ style compilation
// https://github.com/DrAl-HFS/MKF.git
// (c) Project Contributors Jan-June 2019

#ifndef MKF_CUDA_CU
#define MKF_CUDA_CU // supress header "multiple definition" glitch
#endif

#include "mkfCUDA.h"

#ifdef MKF_CUDA_CU
#undef MKF_CUDA_CU // header glitch supression done
#endif

// Wide counter for atomicAdd (nvcc dislikes size_t)
typedef unsigned long long CUACount;


/***/

// CUDA kernels and wrappers

#define BLKS 5
#define BLKD (1<<BLKS)
#define BLKM BLKD-1
#define BINS (1<<8)
//define BLKN 1024/BLKD

__global__ void vThresh8 (uint r[], const float f[], const size_t n, const BinMapF32 mc)
{
   const size_t i= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ uint z[BLKD];
   if (i < n)
   {
      const int j= i & BLKM;
      const int k= i & 0x7; // j & 0x7
      const int d= 1 + (f[i] > mc.t[0]) - (f[i] < mc.t[0]);
      z[j]= ((mc.m >> d) & 0x1) << k; // smaller shift faster ?

      __syncthreads();

      if (0 == k)
      {  // j : { 0, 8, 16, 24 } 4P, 7I
         for (int l=1; l<8; l++) { z[j]|= z[j+l]; }

         __syncthreads();

         if (0 == j)
         {
            r[i>>BLKS]= ( z[0] << 0 ) | ( z[8] << 8 ) | ( z[16] << 16 ) | ( z[24] << 24 );
         }
      }
   }
} // vThresh8

__global__ void vThresh32 (uint r[], const float f[], const size_t n, const BinMapF32 mc)
{
   const size_t i= blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ uint z[BLKD];
   if (i < n)
   {
      const int j= i & BLKM;
      const int d= 1 + (f[i] > mc.t[0]) - (f[i] < mc.t[0]);
      z[j]= ((mc.m >> d) & 0x1) << j; // assume "barrel" shifter

      __syncthreads();

      if (0 == (j & 0x3))
      {  // j : { 0, 4, 8, 12, 16, 10, 24, 28 } 8P 3I
         for (int l=1; l<4; l++) { z[j]|= z[j+l]; }

         __syncthreads();

         //if (0 == j) { r[i>>BLKS]= z[0] | z[4] | z[8] | z[12] | z[16] | z[20] | z[24] | z[28]; }
         if (0 == (j & 0xF))
         {  // j : { 0, 16 } 2P 3I
            for (int l=4; l<16; l+=4) { z[j]|= z[j+l]; }

            __syncthreads();

            if (0 == j) { r[i>>BLKS]= z[0] | z[16]; }
         }
      }
   }
} // vThresh32

#define CHUNK_SHIFT (5)
#define CHUNK_SIZE (1<<CHUNK_SHIFT)
#define CHUNK_MASK (CHUNK_SIZE-1)

__device__ void loadChunkSh0
(
   size_t bufChunk[4],  // chunk buffer
   const uint * pR0,    // Location within row of first -
   const uint * pR1,    //  - and second planes
   const int rowStride  // stride between successive rows within each plane
)
{  // vect
   bufChunk[0]= pR0[0];
   bufChunk[1]= pR0[rowStride];
   bufChunk[2]= pR1[0];
   bufChunk[3]= pR1[rowStride];
} // loadChunkSh0

__device__ void loadChunkSh1
(
   size_t bufChunk[4],  // chunk buffer
   const uint * pR0,    // Location within row of first -
   const uint * pR1,    //  - and second planes
   const int rowStride  // stride between successive rows within each plane
)
{  // vect
   bufChunk[0] |= (pR0[0] << 1);
   bufChunk[1] |= (pR0[rowStride] << 1);
   bufChunk[2] |= (pR1[0] << 1);
   bufChunk[3] |= (pR1[rowStride] << 1);
} // loadChunkSh1

__device__ void ap4x2xN (uint bpfd[BINS], size_t bufChunk[4], const int n)
{
   for (int i= 0; i < n; i++)
   {
      unsigned char bp=  (bufChunk[0] & 0x3) |
               ((bufChunk[1] & 0x3) << 2) |
               ((bufChunk[2] & 0x3) << 4) |
               ((bufChunk[3] & 0x3) << 6);
      bpfd[ bp ]+= 1;
      bufChunk[0] >>= 1;
      bufChunk[1] >>= 1;
      bufChunk[2] >>= 1;
      bufChunk[3] >>= 1;
   }
} // ap4x2xN

__device__ uint lognu (int id, uint u[], const int n)
{
   uint t=0;
   printf("(%d: ", id);
   for (int i= 0; i < n; i++)
   {
      t+= u[i];
      if (0!=u[i]) { printf("%d:%u ",i,u[i]); }
   }
   printf(":%d %u|%u)\n", id, t, t/63);
   return(t);
} // lognu

__device__ void addRowBPFD
(
   uint        bpfd[BINS], // result pattern distribution
   const uint  * pRow[2],
   const int   rowStride,
   const int   n    // Number of single bit elements packed in row
)
{  // seq
   //uint dbg[4]={0,0,0,0};
   int m, k, i;
   size_t bufChunk[4]= { 0,0,0,0 };

   //dbg[3]= lognu(bpfd,256);
   // First chunk of n bits yields n-1 patterns
   loadChunkSh0(bufChunk, pRow[0]+0, pRow[1]+0, rowStride);
   k= MIN(CHUNK_SIZE-1, n-1); //dbg[0]+= k;
   ap4x2xN(bpfd, bufChunk, k);
   // Subsequent whole chunks yield n patterns
   i= 0;
   m= n>>CHUNK_SHIFT;
   while (++i < m)
   {
      loadChunkSh1(bufChunk, pRow[0]+i, pRow[1]+i, rowStride);
      ap4x2xN(bpfd, bufChunk, CHUNK_SIZE); //dbg[1]+= CHUNK_SIZE;
   }
   // Check for residual bits < CHUNK_SIZE
   k= n & CHUNK_MASK;
   if (k > 0)
   {
      loadChunkSh1(bufChunk, pRow[0]+i, pRow[1]+i, rowStride);
      ap4x2xN(bpfd, bufChunk, k); //dbg[2]+= k;
   }
   //printf(" dbg: %u,%u,%u; bpfd: %u+%u\n",dbg[0],dbg[1],dbg[2], dbg[3],lognu(bpfd,256)-dbg[3]);
} // addRowBPFD

__global__ void addPlaneBPFD (CUACount rBPFD[256], const uint * pPln0, const uint * pPln1, const int rowStride, const int defW, const int defH)
{
   const size_t i= blockIdx.x * blockDim.x + threadIdx.x; // ???
//   __shared__ uint bpfd[BINS][BLKD]; // C-row-major (lexicographic) memory order. 32KB
   __shared__ uint bpfd[BINS*BLKD]; // 32KB

   //if (blockDim.x > BLKD) { printf("ERROR: addPlaneBPFD() - blockDim=%d", blockDim.x); return; }
   //else { printf(" - blockDim=%d,%d,%d\n", blockDim.x, blockDim.y, blockDim.z); }
   if (i < defH)
   {
      const uint * pRow[2]= { pPln0 + i*rowStride, pPln1 + i*rowStride };
      const int r= i & BLKM;

      for (int k= r; k < BINS; k+= BLKD)
      {  // (transposed zeroing for write coalescing)
         //for (int j= 0; j < BLKD; j++) { bpfd[k][j]= 0; }
         for (int j= 0; j < BLKD; j++) { bpfd[j*BINS+k]= 0; }
      }

      //addRowBPFD(&(bpfd[0][r]), pRow, rowStride, defW);
      addRowBPFD(bpfd+r*BINS, pRow, rowStride, defW);

      //if (0 == r) { for (int k= 0; k < BLKM; k++) { lognu(k,bpfd+k*BINS,BINS); } }
      __syncthreads(); // Perhaps unneccessary (?) - control flow divergence not possible...

      for (int k= r; k < BINS; k+= BLKD)
      {  // (transposed reduction for read coalescing)
         CUACount t= 0;
         //for (int j= 0; j < BLKD; j++) { t+= bpfd[k][j]; }
         for (int j= 0; j < BLKD; j++) { t+= bpfd[j*BINS+k]; }
         atomicAdd( rBPFD+k, t );
      }
   }
} // addPlaneBPFD


/***/

hipError_t ctuErr (hipError_t *pE, const char *s)
{
   hipError_t e;
   if (NULL == pE) { e= hipGetLastError(); } else { e= *pE; }
   if (0 != e)
   {
      ERROR("%s - r=%d -> %s\n", s, e, hipGetErrorName(e));
   }
   return(e);
} // ctuErr

extern "C" int mkfCUDAGetBPFDSimple (Context *pC, const int def[3], const BinMapF32 *pBM)
{
   hipError_t r;

   if (pC->pHF)
   {
      //r= hipMemcpy(pC->pDU, pC->pHU, pC->bytesU, hipMemcpyHostToDevice);

      if (NULL == pC->pDF)
      {
         r= hipMalloc(&(pC->pDF), pC->bytesF);
         ctuErr(&r, "hipMalloc()");
      }
      if (pC->pDF)
      {
         r= hipMemcpy(pC->pDF, pC->pHF, pC->bytesF, hipMemcpyHostToDevice);
         ctuErr(&r, "hipMemcpy()");
      }

      if (NULL == pC->pDU)
      {
         r= hipMalloc(&(pC->pDU), pC->bytesU);
         ctuErr(&r, "hipMalloc()");
      }

      if (pC->pDF && pC->pDU)
      {
         int blkD= BLKD;//256;
         int nBlk;

         if (pC->pDZ) { hipMemset(pC->pDZ, 0, pC->bytesZ); }
         if (pC->nF <= blkD) { blkD= BLKD; }
         nBlk= (pC->nF + blkD-1) / blkD;
         // CAVEAT! Treated as 1D
         vThresh32<<<nBlk,blkD>>>(pC->pDU, pC->pDF, pC->nF, *pBM);
         ctuErr(NULL, "vThresh32()");
         hipDeviceSynchronize();

         if (pC->pHU)
         {
            LOG("hipMemcpy(%p, %p, %u)\n", pC->pHU, pC->pDU, pC->bytesU);
            r= hipMemcpy(pC->pHU, pC->pDU, pC->bytesU, hipMemcpyDeviceToHost);
            ctuErr(NULL, "{vThresh32+} hipMemcpy()");
         }

         if (pC->bytesZ> 0)
         {
            if (NULL == pC->pDZ)
            {
               r= hipMalloc(&(pC->pDZ), pC->bytesZ);
               ctuErr(&r, "hipMalloc()");
               hipMemset(pC->pDZ, 0, pC->bytesZ);
            }
            if (NULL == pC->pHZ)
            {
               r= hipHostMalloc(&(pC->pHZ), pC->bytesZ);
               ctuErr(&r, "hipMalloc()");
            }
            if (pC->pDZ)
            {
               //size_t bpdBytes= 256*sizeof(uint);
               //if ((pC->pDZ) && (pC->bytesZ >= bpdBytes))
               CUACount *pBPFD= (CUACount*)(pC->pDZ);
               const int rowStride= def[0] / 32;
               const int nRowPairs= def[1]-1;
               const int nPlanePairs= def[2]-1;
               const int planeStride= def[1] * rowStride;

               //if (nRowPairs <= blkD) {
               blkD= BLKD;
               nBlk= (nRowPairs + blkD-1) / blkD;

               for (int i= 0; i < nPlanePairs; i++)
               {
                  const uint *pP0= pC->pDU + i * planeStride;
                  const uint *pP1= pC->pDU + (i+1) * planeStride;
                  addPlaneBPFD<<<nBlk,blkD>>>(pBPFD, pP0, pP1, rowStride, def[0], nRowPairs);
                  if (0 != ctuErr(NULL, "addPlane"))
                  { LOG(" .. <<<%d,%d>>>(%p, %p, %p ..)", nRowPairs, BLKD, pBPFD, pP0, pP1); }
               }
               hipDeviceSynchronize();
               if (pC->pHZ)
               {
                  r= hipMemcpy(pC->pHZ, pC->pDZ, pC->bytesZ, hipMemcpyDeviceToHost);
                  ctuErr(&r, "{addPlane+} hipMemcpy()");
               }
            }
         }
      }
   }

   return(1); //0 == r);
} // mkfCUDAGetBPFDSimple


#ifdef MKF_CUDA_MAIN

#include "geomHacks.h"
#include "mkfUtil.h"

int buffAlloc (Context *pC, const int def[3], const int blkZ)
{
   int vol= def[0] * def[1] * def[2];

   pC->nF= vol;
   pC->bytesF= sizeof(*(pC->pHF)) * pC->nF;
   pC->nU= BITS_TO_WRDSH(vol,5);
   pC->bytesU= sizeof(*(pC->pHU)) * pC->nU;
   pC->nZ= blkZ * 256;
   pC->bytesZ= 8 * pC->nZ; // void * sizeof(*(pC->pHZ))

   LOG("F: %zu -> %zu Bytes\nU: %zu -> %zu Bytes\n", pC->nF, pC->bytesF, pC->nU, pC->bytesU);

   return cuBuffAlloc(pC,vol);
} // buffAlloc


static const char gSepCh[2]={' ','\n'};

void dumpF (const float f[], const int n, const int wrap)
{
   int i=0;
   while (i<n)
   {
      int k= i + wrap;
      if (k > n) { k= n; }
      for (int j= i; j < k; j++) { LOG("%G%c", f[j], gSepCh[(j+1)==k]); }
      i= k;
   }
} // dumpF

void dumpUX (const uint u[], const int n, const int wrap)
{
   int i=0;
   while (i<n)
   {
      int k= i + wrap;
      if (k > n) { k= n; }
      for (int j= i; j < k; j++) { LOG("%08X%c", u[j], gSepCh[(j+1)==k]); }
      i= k;
   }
} // dumpUX

size_t bitCountNU32 (U32 u[], const int n)
{
   size_t t= 0;
   for (int i= 0; i<n; i++) { t+= bitCountZ(u[i]); }
   return(t);
} // bitCountNU32

void mkft (Context *pC, const int def[3], U8 id, const float radius)
{
   const char *name[2]={"ball","box"};
   BinMapF32 bmc;
   float vr, fracR= radius / def[1];
   size_t t;
   int n;

   switch(id)
   {
      case 1 :
         vr= boxVol(fracR);
         n= genBlock(pC->pHF, def, radius);
         break;
      default :
         id= 0;
         vr= sphereVol(fracR);
         n= genBall(pC->pHF, def, radius);
         break;
   }
   LOG("[%d,%d,%d] %s(%G)->%zu (/%d=%G, ref=%G)\n", def[0], def[1], def[2], name[id], radius, n, pC->nF, (F64)n / pC->nF, vr);
   //dumpF(pC->pHF+n, n, def[0]);
   setBinMapF32(&bmc,">=",0.5);
   LOG("***\nmkfCUDAGetBPFDSimple() - bmc: %f,0x%X\n",bmc.t[0], bmc.m);
   mkfCUDAGetBPFDSimple(pC, def, &bmc);
#if 0
   t= bitCountNU32(pC->pHU, pC->bytesU>>2);
   LOG("bitCountNU32() -> %zu\n", t);
   LOG("%p[%u]:\n",pC->pHU,pC->nU);
   m= def[0] >> BLKS; // def[0] / BLKD;
   n= m * def[1];
   if (n > pC->nU/2) { n= pC->nU/2; }
   while ((m<<1) < 16) { m<<= 1; }
   dumpUX(pC->pHU+2*n, n, m);
   LOG("%s\n","-");
   dumpUX(pC->pHU+3*n, n, m);
#endif
   if (pC->pHZ)
   {
      const size_t *pBPFD= (size_t*)pC->pHZ;
      LOG("\tvolFrac=%G chiEP=%G\n", volFrac(pBPFD), chiEP3(pBPFD));

      for (int i= 0; i<256; i++)
      {
         if (pBPFD[i] > 0) { LOG("[%d]=%u\n", i, pBPFD[i]); }
      }
   }
} // mkft

#if 0
__global__ void vAddB (float r[], const float a[], const float b[], const int n)
{
   int i= blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) { r[i]= a[i] + b[i]; }
} // vAddB

void sanityTest (Context *pC)
{
   const int n= 1024;
   int i, e=0;
   for (i=0; i<n; i++) { pC->pHF[i]= i; pC->pHF[2*n - (1+i)]= 1+i; }
   hipMemcpy(pC->pDF, pC->pHF, 2*n*sizeof(pC->pHF[0]), hipMemcpyHostToDevice); ctuErr(NULL, "hipMemcpy 1");
   vAddB<<<8,128>>>(pC->pDF+2*n, pC->pDF+0, pC->pDF+n, n);
   hipMemcpy(pC->pHF+2*n, pC->pDF+2*n, n*sizeof(pC->pHF[0]), hipMemcpyDeviceToHost); ctuErr(NULL, "hipMemcpy 2");

   i= 2 * n;
   LOG("sanityTest() - vAddB() - [%d]=%G", i, pC->pHF[i]);
   for ( ; i < (3*n)-1; i++)
   {
      if (pC->pHF[i] != n) { ++e; LOG(" [%d]=%G", i, pC->pHF[i]); }
   }
   LOG(" [%d]=%G\n", i, pC->pHF[i]);

   printf("*e=%d*\n", e);
} // sanityTest();
#endif

int main (int argc, char *argv[])
{
   const int def[3]= {64,64,128};
   Context cux={0};

   if (buffAlloc(&cux, def, 1))
   {
      //sanityTest(&cux);
      mkft(&cux, def, 0, 0.5*def[1] - 2.5);
      cuBuffRelease(&cux);
   }
   hipDeviceReset();
} // main

#endif // MKF_CUDA_MAIN
